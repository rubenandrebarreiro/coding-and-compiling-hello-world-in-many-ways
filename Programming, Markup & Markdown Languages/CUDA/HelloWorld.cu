/**
*
* Hello World in Many Programming Languages!!!
*
* Authors:
* @author Ruben Andre Barreiro
*
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world() {
    // Doesn't do anything here
}

int main() {
   
    // Invoking CUDA Kernel with 1 thread, on 1 block
    hello_world<<<1,1>>>();

    // Prints "Hello World!!!" to the terminal window
    printf("Hello World!!!\n");
    
    return EXIT_SUCCESS;
}
